#include "hip/hip_runtime.h"
#include "cuda_macros.h"
#include "active_set_buffers.h"

#define BLOCK_DIM_X 128
#define GRID_DIM_X 64

#define BLOCK_DIM_BATCH_X 32
#define BLOCK_DIM_BATCH_Y 32

#define BLOCK_DIM_NORM_X 128
#define BLOCK_DIM_NORM_Y 1

#define MAT_IJ_TO_LINEAR(i, j, dim) ((i) + (j)*(dim))

extern "C" void construct_active_set_buffers(ActiveSetBuffers *buffers, int dim_input, int dim_target, int max_active) {
  // assign params
  buffers->max_active = max_active;
  buffers->num_active = 0;
  buffers->dim_input = dim_input;
  buffers->dim_target = dim_target;  

  // allocate buffers
  cudaSafeCall(hipMalloc((void**)&(buffers->active_inputs), dim_input * max_active * sizeof(float)));
  cudaSafeCall(hipMalloc((void**)&(buffers->active_targets), dim_target * max_active * sizeof(float)));
  cudaSafeCall(hipMalloc((void**)&(buffers->active_kernel_matrix), max_active * max_active * sizeof(float)));

  // set kernel matrix to all zeros
  cudaSafeCall(hipMemset(buffers->active_targets, 0, max_active * sizeof(float)));
  cudaSafeCall(hipMemset(buffers->active_kernel_matrix, 0, max_active * max_active * sizeof(float)));
}

extern "C" void free_active_set_buffers(ActiveSetBuffers *buffers) {
  // free everything
  cudaSafeCall(hipFree(buffers->active_inputs));
  cudaSafeCall(hipFree(buffers->active_targets));
  cudaSafeCall(hipFree(buffers->active_kernel_matrix));
}

__device__ float exponential_kernel(float* x, float* y, int dim, int sigma)
{
  float sum = 0;
  for (int i = 0; i < dim; i++) {
    sum += __fmul_rn(__fadd_rn(x[i], -y[i]), __fadd_rn(x[i], -y[i]));
    //    printf("sum %f\n", sum);
  }
  return __expf(-sum / (2 * sigma));
}

__global__ void compute_kernel_vector_kernel(float* active_inputs, float* all_inputs, float* kernel_vector, int index, float sigma, int dim_input, int num_pts, int num_active, int max_active)
{
  float local_new_input[MAX_DIM_INPUT];
  float local_active_input[MAX_DIM_INPUT];

  int global_x = threadIdx.x + blockDim.x * blockIdx.x;
  float kernel_val = 0.0f;

  if (global_x >= max_active)
    return;

  // float test = all_inputs[1];
  // if (threadIdx.x == 0 && blockIdx.x == 0)
  //   printf("Test kernel %f\n", test);

  __syncthreads();
  if (global_x < num_active) {
    // read new input into local memory
    for (int i = 0; i < dim_input; i++) {
      local_new_input[i] = all_inputs[index + i*num_pts];
      //      printf("KV New %d %d %f \n", i, index, local_new_input[i]);
    }
    // coalesced read of active input to compute kernel with
    for (int i = 0; i < dim_input; i++) {
      local_active_input[i] = active_inputs[global_x + i*num_pts];
      //      printf("Active %d %d %f \n", i, global_x, local_active_input[i]);
    }

    kernel_val = exponential_kernel(local_new_input, local_active_input, dim_input, sigma);
    //    printf("Kernel val %d %f\n", index, kernel_val/*, local_new_input[0], local_new_input[1], local_active_input[0], local_active_input[1]*/);
  }

  // coalesced value write to vector
  __syncthreads();
  kernel_vector[global_x] = kernel_val;
}

extern "C" void compute_kernel_vector(ActiveSetBuffers *active_buffers, MaxSubsetBuffers *subset_buffers, int index, float* kernel_vector, GaussianProcessHyperparams hypers)
{
  dim3 block_dim(BLOCK_DIM_X, 1, 1);
  dim3 grid_dim(ceilf((float)(active_buffers->num_active)/(float)(block_dim.x)), 1, 1);

  cudaSafeCall((compute_kernel_vector_kernel<<<grid_dim, block_dim>>>(active_buffers->active_inputs, subset_buffers->inputs, kernel_vector, index, hypers.sigma, active_buffers->dim_input, subset_buffers->num_pts, active_buffers->num_active, active_buffers->max_active)));
}

__global__ void compute_kernel_vector_batch_kernel(float* active_inputs, float* all_inputs, float* kernel_vectors, int index, int batch_size, float sigma, int dim_input, int num_pts, int num_active, int max_active)
{
  float local_new_input[MAX_DIM_INPUT];
  float local_active_input[MAX_DIM_INPUT];

  int global_x = threadIdx.x + blockDim.x * blockIdx.x; // active point to grab
  int global_y = threadIdx.y + blockDim.y * blockIdx.y; // point to operate on (offset from index)
  float kernel_val = 0.0f;

  if (global_x >= max_active || global_y >= num_pts - index || global_y >= batch_size)
    return;

  __syncthreads();
  if (global_x < num_active) {
    // read new input into local memory
    for (int i = 0; i < dim_input; i++) {
      local_new_input[i] = all_inputs[global_y + index + i*num_pts];
      //      printf("KV New %d %d %f \n", i, index, local_new_input[i]);
    }
    // coalesced read of active input to compute kernel with
    for (int i = 0; i < dim_input; i++) {
      local_active_input[i] = active_inputs[global_x + i*max_active];
      //printf("Active %d %d %f \n", i, global_x, local_active_input[i]);
    }

    kernel_val = exponential_kernel(local_new_input, local_active_input, dim_input, sigma);
    //    printf("Kernel val %d %d %d %f\n", num_active, global_x, global_y, kernel_val/*, local_new_input[0], local_new_input[1], local_active_input[0], local_active_input[1]*/);
  }

  // coalesced value write to vector
  __syncthreads();
  kernel_vectors[global_x + global_y*max_active] = kernel_val;
}

extern "C" void compute_kernel_vector_batch(ActiveSetBuffers *active_buffers, MaxSubsetBuffers* subset_buffers, int index, int batch_size, float* kernel_vectors, GaussianProcessHyperparams hypers)
{
  // x corresponds to the active point to compute the kernel with
  // y corresponds to the query point
  dim3 block_dim(BLOCK_DIM_BATCH_X, BLOCK_DIM_BATCH_Y, 1);
  dim3 grid_dim(ceilf((float)(active_buffers->num_active)/(float)(block_dim.x)),
		ceilf((float)(batch_size)/(float)(block_dim.y)),
		1);

  cudaSafeCall((compute_kernel_vector_batch_kernel<<<grid_dim, block_dim>>>(active_buffers->active_inputs, subset_buffers->inputs, kernel_vectors, index, batch_size, hypers.sigma, active_buffers->dim_input, subset_buffers->num_pts, active_buffers->num_active, active_buffers->max_active)));
}

__global__ void update_kernel_matrix_kernel(float* kernel_matrix, float* active_inputs, float* active_targets, float* all_inputs, float* all_targets, float beta, float sigma, int* g_index, int dim_input, int dim_target, int num_pts, int num_active, int max_active)
{
  // parameters
  __shared__ int segment_size;
  __shared__ int index;

  float local_new_input[MAX_DIM_INPUT];
  float local_active_input[MAX_DIM_INPUT];
  float local_new_target[MAX_DIM_INPUT];

  // read global variables into shared memory
  if (threadIdx.x == 0) {
    segment_size = max((int)ceilf((float)(num_active+1)/(float)GRID_DIM_X), 1);
    index = g_index[0];
  }

  int global_x = 0;
  float kernel = 0.0f;

  __syncthreads();
  for (int i = 0; i * blockDim.x < segment_size; i++) {
    global_x = threadIdx.x + i * blockDim.x + segment_size * blockIdx.x;

    // fetch new data from global menory
    for (int j = 0; j < dim_input; j++) {
      local_new_input[j] = all_inputs[index + j*num_pts];
    } 
    for (int j = 0; j < dim_target; j++) {
      local_new_target[j] = all_targets[index + j*num_pts];
    }

    // fetch active points from global memory
    if (global_x < segment_size * (blockIdx.x + 1) && global_x < num_active) {    
      for (int j = 0; j < dim_input; j++) {
    	local_active_input[j] = active_inputs[global_x + j*max_active];
      }
      
      kernel = exponential_kernel(local_new_input, local_active_input, dim_input, sigma);
    }

    // coalesced write to new column and row
    __syncthreads();
    if (global_x < segment_size * (blockIdx.x + 1) && global_x < num_active) {
      kernel_matrix[MAT_IJ_TO_LINEAR(global_x, num_active, max_active)] = kernel;
      kernel_matrix[MAT_IJ_TO_LINEAR(num_active, global_x, max_active)] = kernel;   
    }

    // coalesced write to active inputs
    __syncthreads();
    if (i == 0 && global_x < dim_input && global_x < segment_size * (blockIdx.x + 1)) {
      active_inputs[num_active + global_x*max_active] = local_new_input[global_x];
      //      printf("new input %d %d %f\n", num_active, global_x, local_new_input[global_x]);
    }
      
    // coalesced write to active targets
    __syncthreads();
    if (i == 0 && global_x < dim_target && global_x < segment_size * (blockIdx.x + 1)) {
      active_targets[num_active + global_x*max_active] = local_new_target[global_x];
      //      printf("new target %d %f\n", global_x, local_new_target[global_x]);
    }
      
    // write diagonal term
    __syncthreads();
    if (i == 0 && global_x == 0) {
      float diag_val = exponential_kernel(local_new_input, local_new_input, dim_input, sigma);
      kernel_matrix[MAT_IJ_TO_LINEAR(num_active, num_active, max_active)] = diag_val + beta;
      //      printf("new diag %d %d %f\n", global_x, MAT_IJ_TO_LINEAR(num_active, num_active, max_active),  kernel_matrix[MAT_IJ_TO_LINEAR(num_active, num_active, max_active)]);
    }
    __syncthreads();
  }
}

extern "C" void update_active_set_buffers(ActiveSetBuffers *active_buffers, MaxSubsetBuffers *subset_buffers, GaussianProcessHyperparams hypers) {

  int dim_input = subset_buffers->dim_input;
  int dim_target = subset_buffers->dim_target;
  if (dim_input > MAX_DIM_INPUT) {
    printf("Error: Input is too high dimensional for update. Aborting...");
    return;
  }
  if (dim_target > MAX_DIM_INPUT) {
    printf("Error: Target is too high dimensional for update. Aborting...");
    return;
  }

  dim3 block_dim(BLOCK_DIM_X, 1, 1);
  dim3 grid_dim(GRID_DIM_X, 1, 1);

  cudaSafeCall((update_kernel_matrix_kernel<<<grid_dim, block_dim>>>(active_buffers->active_kernel_matrix,
								     active_buffers->active_inputs,
								     active_buffers->active_targets,
								     subset_buffers->inputs,
								     subset_buffers->targets,
								     hypers.beta, hypers.sigma,
								     subset_buffers->d_next_index, 
								     dim_input, dim_target,
								     subset_buffers->num_pts,
								     active_buffers->num_active,
								     active_buffers->max_active))); 
  active_buffers->num_active++;
}

__global__ void norm_columns_kernel(float* A, float* x, int m, int n, int lda)
{
  // max score for each thread
  __shared__ float s_sums[BLOCK_DIM_BATCH_X * BLOCK_DIM_BATCH_Y];

  // parameters
  __shared__ int segment_size;
  __shared__ int s_lda;

  if (threadIdx.x == 0 && threadIdx.y == 0) {
    segment_size = m;
    s_lda = lda;
  }

  // initialize scores and count
  int local_x = threadIdx.x + blockDim.x*threadIdx.y;
  s_sums[local_x] = 0;
  __syncthreads();

  int global_x = 0;
  int global_y = threadIdx.y + blockIdx.y*blockDim.y;
  float val = 0.0f;

  // keep reading the values and squaring them
  for (int i = 0; i * blockDim.x < segment_size; i++) {
    global_x = threadIdx.x + i * blockDim.x;
    
    // read from global memory
    __syncthreads();
    if (global_x < segment_size && global_y < n) {
      val = A[global_x + s_lda * global_y];
      //      printf("Read %f at %d, %d\n", val, global_x, global_y); 
      s_sums[local_x] += val * val;
    }
  }

  // reduce the squared sum
  global_x = threadIdx.x;
  for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
    __syncthreads();
    if (global_x < stride && (global_x + stride) < blockDim.x) {
      s_sums[global_x + blockDim.x * threadIdx.y] += s_sums[global_x + blockDim.x * threadIdx.y + stride];
    }
  }

  // write result to global x vector
  __syncthreads();
  if (threadIdx.x == 0) {
    //    printf("Sig at %d: %f\n", threadIdx.y, s_sums[blockDim.x * threadIdx.y]);
    x[threadIdx.y + blockIdx.y * blockDim.y] = s_sums[blockDim.x * threadIdx.y];
  }
}

// square norm the columns of A, store in x
extern "C" void norm_columns(float* A, float* x, int m, int n, int lda)
{
  // y for each column of the matrix
  dim3 block_dim(BLOCK_DIM_NORM_X, BLOCK_DIM_NORM_Y, 1);
  dim3 grid_dim(1,
		ceilf((float)(n)/(float)(block_dim.y)),
		1);

  cudaSafeCall((norm_columns_kernel<<<grid_dim, block_dim>>>(A, x, m, n, lda)));

}
