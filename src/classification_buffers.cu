#include "cuda_macros.h"
#include "classification_buffers.h"

extern "C" void construct_classification_buffers(ClassificationBuffers *buffers, int num_pts) {
  // assign params
  buffers->num_pts = num_pts;
  
  // allocate buffers
  cudaSafeCall(hipMalloc((void**)buffers->above, num_pts * sizeof(unsigned char)));
  cudaSafeCall(hipMalloc((void**)buffers->below, num_pts * sizeof(unsigned char)));

  // set all to 0 (all points are initially undetermined
  cudaSafeCall(hipMemset((void**)buffers->above, 0, num_pts * sizeof(unsigned char)));  
  cudaSafeCall(hipMemset((void**)buffers->below, 0, num_pts * sizeof(unsigned char)));  
}

extern "C" void free_classification_buffers(ClassificationBuffers *buffers) {
  // free everything
  cudaSafeCall(hipFree(buffers->above));
  cudaSafeCall(hipFree(buffers->below));
}

