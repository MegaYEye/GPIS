#include "cuda_macros.h"
#include "classification_buffers.h"

extern "C" void construct_classification_buffers(ClassificationBuffers *buffers, int num_pts) {
  // assign params
  buffers->num_pts = num_pts;
  
  // allocate buffers
  cudaSafeCall(hipMalloc((void**)buffers->upper, num_pts * sizeof(unsigned char)));
  cudaSafeCall(hipMalloc((void**)buffers->lower, num_pts * sizeof(unsigned char)));

  // set all to 0 (all points are initially undetermined
  cudaSafeCall(hipMemset((void**)buffers->upper, 0, num_pts * sizeof(unsigned char)));  
  cudaSafeCall(hipMemset((void**)buffers->lower, 0, num_pts * sizeof(unsigned char)));  
}

extern "C" void free_classification_buffers(ClassificationBuffers *buffers) {
  // free everything
  cudaSafeCall(hipFree(buffers->upper));
  cudaSafeCall(hipFree(buffers->lower));
}

