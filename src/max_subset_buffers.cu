#include "cuda_macros.h"
#include "max_subset_buffers.h"

extern "C" void construct_max_subset_buffers(MaxSubsetBuffers *buffers, float* input_points, float* target_points, int dim_input, int dim_target, int num_pts) {
  // assign params
  buffers->dim_input = dim_input;
  buffers->dim_target = dim_target;
  buffers->num_pts = num_pts;
  
  // allocate buffers
  cudaSafeCall(hipMalloc((void**)buffers->inputs, dim_input * num_pts * sizeof(float)));
  cudaSafeCall(hipMalloc((void**)buffers->targets, dim_target * num_pts * sizeof(float)));
  cudaSafeCall(hipMalloc((void**)buffers->active, num_pts * sizeof(unsigned char)));
  cudaSafeCall(hipMalloc((void**)buffers->scores, num_pts * sizeof(float)));

  // set buffs
  cudaSafeCall(hipMemcpy((void**)(buffers->inputs), input_points, dim_input * num_pts * sizeof(float), hipMemcpyHostToDevice));  
  cudaSafeCall(hipMemcpy((void**)(buffers->targets), target_points, dim_target * num_pts * sizeof(float), hipMemcpyHostToDevice));  

  // set all active to 0 initially
  cudaSafeCall(hipMemset((void**)buffers->active, 0, num_pts * sizeof(unsigned char)));  
}

extern "C" void activate_max_subset_buffers(MaxSubsetBuffers* buffers, int index) {
  cudaSafeCall(hipMemset((void**)(buffers->active + index), 1, sizeof(unsigned char)));
}

extern "C" void free_max_subset_buffers(MaxSubsetBuffers *buffers) {
  // free everything
  cudaSafeCall(hipFree(buffers->inputs));
  cudaSafeCall(hipFree(buffers->targets));
  cudaSafeCall(hipFree(buffers->active));
  cudaSafeCall(hipFree(buffers->scores));
}

